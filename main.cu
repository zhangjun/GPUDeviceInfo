/*******************************************
   Auther        : Jun Zhang
   Email         : ewalker.zj@gmail.com
   Last Modified : 2015-04-01 00:26
   Filename      : main.cu
   Destription   : 
*******************************************/
#include<stdio.h>

void getDeviceInfo()
{
	int gpu_id;
	hipGetDevice(&gpu_id);
	hipDeviceProp_t gpu_dev;
	hipGetDeviceProperties(&gpu_dev, gpu_id);

	size_t total_mem, avail_mem;
	checkCudaErrors(hipMemGetInfo(&avail_mem, &total_mem));
//	int numGPUCores = getGPUCoresNum();
	printf("GPU %s: avail_mem is %d MB, total_mem is %d MB\n", gpu_dev.name, avail_mem/1024/1024, total_mem/1024/1024);
}

int main()
{
	getDeviceInfo();
	return 0;
}
